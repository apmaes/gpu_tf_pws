#include "hip/hip_runtime.h"
/*
	tf-PWS stacking with CUDA
	Xiangfang Zeng Dept. of Geoscience, UW-Madison 
	zengxf@geology.wisc.edu

	version 1.  20150506
	version 2.  20150508
		    reduce many hipMemcpy between host and device
	version 3.  20150511
       	            fix problem in copy_complex_matrix_shift when npts>1024(MAT_thread).
	version 4.  20150511
		    try zero-copy memory.  speed-up ??? 
	version 5.  use page locked/pinned memory, it is better than zero-copy memory 
Ref: 		    
	    tf-pws: Schimmel M. and J. Gallart, 2007, Frequency-dependent phase coherence for noise suppression in seismic array data. J. Geophys. Res. 112, B04303
    	pws to LFE: Thurber et al., 2014 Phase-weighted stacking applied to low-frequency earthquakes. Bull. Seismol. Soc. Am., 104(5), 2567-2572

	sac.h and sacio.c were developed by Dr Lupei Zhu, http://www.eas.slu.edu/People/LZhu/home.html
   
   */
#include <hipfft/hipfft.h>
#include <time.h>
#include "sacio.c"
#define NMAX 1024
#define MAX 8388608
#define MAX_TH 1024
#define MAX_BK 65536 
//kernel functions
__global__ void sum_along_col(hipfftComplex *sum,hipfftComplex *s,int n)
{
	int i;
	int idx=threadIdx.x+blockIdx.x*blockDim.x;
	while(idx<n)
	{
		sum[idx].x=0.0;
		sum[idx].y=0.0;
		for(i=0;i<n;i++)
		{
			sum[idx].x = sum[idx].x + s[idx*n+i].x;
			sum[idx].y = sum[idx].y + s[idx*n+i].y;
		}
		if(hipCabsf(sum[idx])){sum[idx].x=0.0;sum[idx].y=0.0;}
	}
}
__global__ void copy_complex_vector(hipfftComplex *a,hipfftComplex *b,int n)
{
	int idx=threadIdx.x+blockIdx.x*blockDim.x;
	while(idx<n)
	{
                a[idx].x=b[idx].x;
                a[idx].y=b[idx].y;
                idx=idx+blockDim.x*gridDim.x;
	}
}
__global__ void copy_complex_vector_shift(hipfftComplex *a,hipfftComplex *b,int n,int shift)
{
	int idx=threadIdx.x+blockIdx.x*blockDim.x;
	int idy=threadIdx.x+blockIdx.x*blockDim.x+shift;
	if(idy>=n) idy=idy-n;
	while(idx<n)
	{
		
                a[idx].x=b[idy].x;
                a[idx].y=b[idy].y;
                idx=idx+blockDim.x*gridDim.x;
	}
}
__global__ void copy_complex_matrix_shift(hipfftComplex *a,hipfftComplex *b,int n)
{
	//b is a new_npts long vector
	//a is a new_npt*new_npts matrix
	int x=threadIdx.x+blockIdx.x*blockDim.x;
	int y=threadIdx.y+blockIdx.y*blockDim.y;
	int idx,idb;
	idx=x+y*n;
	idb=x+y;
	if(idb>=n) idb=idb-n;
                a[idx].x=b[idb].x;
                a[idx].y=b[idb].y;
}
__global__ void copy_complex_matrix(hipfftComplex *a,hipfftComplex *b,int n)
{
	int idx=threadIdx.x+blockIdx.x*blockDim.x;
	while(idx<n*n)
	{
                a[idx].x=b[idx].x;
                a[idx].y=b[idx].y;
                idx=idx+blockDim.x*gridDim.x;
	}
}
__global__ void complex_mul_float_mat(hipfftComplex *a,float *b,int n)
{
	int idx=threadIdx.x+blockIdx.x*blockDim.x;
	while(idx<n*n)
	{
                a[idx].x=a[idx].x*b[idx];
                a[idx].y=a[idx].y*b[idx];
                idx=idx+blockDim.x*gridDim.x;
	}
}
__global__ void update_smat(hipfftComplex *s,hipfftComplex *ft,int n)
{
	int idx=threadIdx.x+blockIdx.x*blockDim.x;
	float abs;
	float tmpx,tmpy;
	while(idx<n*n)
	{
		abs=hipCabsf(s[idx]);
		if(isnan(abs)) abs=1;
                tmpx=(s[idx].x * ft[idx].x - s[idx].y*ft[idx].y)/abs;
                tmpy=(s[idx].y * ft[idx].x + s[idx].x*ft[idx].y)/abs;
                s[idx].x=tmpx;
		s[idx].y=tmpy;
                idx=idx+blockDim.x*gridDim.x;

	}
}
__global__ void sum_smat(hipfftComplex *sum,hipfftComplex *s,int n)
{
	int idx=threadIdx.x+blockIdx.x*blockDim.x;
	while(idx<n*n)
	{
                sum[idx].x=sum[idx].x+s[idx].x;
                sum[idx].y=sum[idx].y+s[idx].y;
                idx=idx+blockDim.x*gridDim.x;
	}
}
//wght is a complex whereas real weight is wght.x
__global__ void mul_wght_mat(hipfftComplex *s,hipfftComplex *w,int n)
{
	int idx=threadIdx.x+blockIdx.x*blockDim.x;
	while(idx<n*n)
	{
                s[idx].x=s[idx].x*w[idx].x;
                s[idx].y=s[idx].y*w[idx].x;
                idx=idx+blockDim.x*gridDim.x;
	}
}
__global__ void pws_wght_mat(hipfftComplex *s,int n,int ntr,int pwr)
{
	int idx=threadIdx.x+blockIdx.x*blockDim.x;
	float abs;
	while(idx<n*n)
	{
		abs=sqrt(s[idx].x*s[idx].x+s[idx].y*s[idx].y);
		s[idx].x=abs*abs/ntr; //when pwr = 2
                idx=idx+blockDim.x*gridDim.x;
	}	
}
////main function and other cpu subroutines 
int main(int ac,char **av)
{
	int nextpow2(int);
	void read_trace_list(char*,float**,int*,int*,float*,float*);

	//n*n matrix operators
	void gaussian_matrix(float*,int,float*,int);
	void pws_wght(hipfftComplex*,int,int,int);
	void complex_mul_float(hipfftComplex *,float*,int);
	void update_s_mat(hipfftComplex*,hipfftComplex*,int);
	void sum_s_mat(hipfftComplex*,hipfftComplex*,int);
	void mul_wght(hipfftComplex*,hipfftComplex*,int);

	
	if(ac!=3)
	{
		fprintf(stderr,"xxx input_list output_sac\nMax %d traces\nnpts is not larger than 1024?2048 (depends on your device)\n",NMAX);
		exit(-1);
	}

	//timers
        clock_t t1,t2,t3,t4,t5,t6,t7;
	double time_cost;       

	int k=2;
	int pwr=2;
	//input data
	int ntrace;
	int new_npts,npts;
	float dt,b;
	float **dat;
	dat=(float**)malloc(sizeof(float*)*NMAX);
	
	int i,j;
	//cufft
	hipfftHandle plan0,plan1,plan2; //plan0: 1 traces, plan1: all input traces batch=nstrace, plan2: each trace tf ifft batch=new_npts/nloop
	//storage spectra of all traces
	hipfftComplex *host_dat;
	//store spectrum of linear-stacking traces
	hipfftComplex *host_stack;
	hipfftComplex *dev_stack;
	//temp variable of spectrum
//	hipfftComplex *host_tmp;
	hipfftComplex *dev_tmp;

	hipfftComplex *dev_dat;
	hipfftResult res;
	//control parameters of s trans. 
	int batch,nloop,shift;
	//for matrix new_npts*new_npts
	int threads,blocks;

	//gaussian matrix
	float df;
	float *freq_vec;
	float *time_vec;
	float *w_mat;
	float *dev_w_mat;
	
	//s matrix
	hipfftComplex *s_mat,*dev_s_mat,*dev_sum_s;
	//ft matrix
	hipfftComplex *ft_mat,*dev_ft_mat;
	float ft_tmp;
	float pi=3.14159;

	//read input data	
	t1=clock();
	read_trace_list(av[1],dat,&ntrace,&npts,&dt,&b);
	fprintf(stderr,"trace no: %d npts: %d dt: %f\n",ntrace,npts,dt);
	t2=clock();
	new_npts=nextpow2(npts);

	//we know threads and blocks
	if(new_npts >= MAX_TH)
		threads=MAX_TH;
	else
		threads=new_npts;
	blocks=new_npts*new_npts/threads;
	if(blocks > MAX_BK) blocks=MAX_BK;
	//set up bk and th in copy_complex_matrix_shift
	int nth=32;
	int nbk=new_npts/nth;
	dim3 th(nth,nth);
	dim3 bk(nbk,nbk);	

	//fprintf(stderr,"blocks %d threads per block %d\n",blocks,threads);
	df=1/((new_npts-1)*dt);
	freq_vec=(float*)malloc(sizeof(float)*new_npts);
	time_vec=(float*)malloc(sizeof(float)*new_npts);
	ft_mat=(hipfftComplex*)malloc(sizeof(hipfftComplex)*new_npts*new_npts);
	hipMalloc((void**)&dev_ft_mat,sizeof(float)*new_npts*new_npts);
	for(i=0;i<new_npts/2;i++)
	{	
		freq_vec[i]=i*df;
		time_vec[i]=i*dt;
	}
	for(;i<new_npts;i++)
	{
		freq_vec[i]=(new_npts-i)*df;
		time_vec[i]=i*dt;
	}
	for(i=0;i<new_npts;i++)
	{
		for(j=0;j<new_npts;j++)
		{

			ft_tmp = freq_vec[j]*time_vec[i];
			ft_mat[j*new_npts+i].y=sin(ft_tmp*2*pi);
			ft_mat[j*new_npts+i].x=cos(ft_tmp*2*pi);
		}
	}
//	fprintf(stderr,"set up ft matrix\n");
	w_mat=(float*)malloc(sizeof(float)*new_npts*new_npts);
	hipMalloc((void**)&dev_w_mat,sizeof(float)*new_npts*new_npts);
	
	t3=clock();
	//cufft of input data
	hipMalloc((void**)&dev_dat,sizeof(hipfftComplex)*ntrace*new_npts);
	hipMalloc((void**)&dev_stack,sizeof(hipfftComplex)*new_npts);
	int n1[1]={new_npts};
	//host_dat=(hipfftComplex*)malloc(sizeof(hipfftComplex)*ntrace*new_npts);

	//pinned 
	hipHostAlloc((void**)&host_dat,sizeof(hipfftComplex)*ntrace*new_npts,hipHostMallocDefault);

	hipMalloc((void**)&dev_tmp,sizeof(float)*2*new_npts);
	//try new one
	//	host_stack=(hipfftComplex*)malloc(sizeof(hipfftComplex)*new_npts);
	//pinned 
	hipHostAlloc((void**)&host_stack,sizeof(hipfftComplex)*ntrace*new_npts,hipHostMallocDefault);
//	hipHostGetDevicePointer(&dev_stack,host_stack,0);
	for(i=0;i<ntrace;i++)
	{
		for(j=0;j<npts;j++)
		{
		if(i!=0)
			host_stack[j].x=host_stack[j].x+dat[i][j];
		else
			host_stack[j].x=dat[i][j];
		
                host_dat[i*new_npts+j].x=dat[i][j];
                host_dat[i*new_npts+j].y=0.0;
		}
		for(;j<new_npts;j++)
		{
                host_dat[i*new_npts+j].x=0.0;
                host_dat[i*new_npts+j].y=0.0;
		}
	}
	for(j=0;j<npts;j++)
	{
		host_stack[j].x=host_stack[j].x/ntrace;
		host_stack[j].y=0.0;
	}	
	for(j=npts;j<new_npts;j++)
	{
		host_stack[j].x=0.0;
                host_stack[j].y=0.0;
	}
	//cpy
//comment it if new zero copy
	hipMemcpy(dev_dat,host_dat,sizeof(hipfftComplex)*ntrace*new_npts,hipMemcpyHostToDevice);
	res=hipfftPlanMany(&plan1,1,n1,NULL,1,new_npts,NULL,1,new_npts,HIPFFT_C2C,ntrace);
	if(res!=0)
	{
	fprintf(stderr,"fail to create plan, error # %d\nCUFFT_INVALID_PLAN = 1\nCUFFT_ALLOC_FAILED = 2\nCUFFT_INVALID_TYPE = 3\nCUFFT_INVALID_VALUE = 4\nCUFFT_INTERNAL_ERROR = 5\nCUFFT_EXEC_FAILED = 6\nCUFFT_SETUP_FAILED = 7\nCUFFT_INVALID_SIZE = 8\nCUFFT_INCOMPLETE_PARAMETER_LIST = 10\n HIPFFT_INVALID_DEVICE = 11\nCUFFT_PARSE_ERROR = 12\nCUFFT_NO_WORKSPACE = 13\n",res);	
	exit(-1);
	}
	hipfftExecC2C(plan1,dev_dat,dev_dat,HIPFFT_FORWARD);
	//hipDeviceSynchronize();
	t4=clock();
	//for test
	//prepare w matrix
	gaussian_matrix(w_mat,k,freq_vec,new_npts);
	//cpy w and ft matrix to gpu
	hipMemcpy(dev_w_mat,w_mat,sizeof(float)*new_npts*new_npts,hipMemcpyHostToDevice);
	hipMemcpy(dev_ft_mat,ft_mat,sizeof(float)*new_npts*new_npts,hipMemcpyHostToDevice);
	//allocate s matrix
	s_mat=(hipfftComplex*)malloc(sizeof(hipfftComplex)*new_npts*new_npts);
	hipMalloc((void**)&dev_s_mat,sizeof(hipfftComplex)*new_npts*new_npts);
	hipMalloc((void**)&dev_sum_s,sizeof(hipfftComplex)*new_npts*new_npts);

	//s transform of each traces
	if(new_npts*new_npts < MAX)
	{
		nloop = 1;
	}	
	else
	{
		nloop = new_npts*new_npts/MAX;
	}
	batch = new_npts/nloop;
	shift=batch*new_npts;
	//allocate plan
	res=hipfftPlanMany(&plan2, 1, n1, NULL, 1, new_npts, NULL, 1, new_npts, HIPFFT_C2C,batch);
	if(res!=0)
	{
	fprintf(stderr,"fail to create stran plan, error # %d\nCUFFT_INVALID_PLAN = 1\nCUFFT_ALLOC_FAILED = 2\nCUFFT_INVALID_TYPE = 3\nCUFFT_INVALID_VALUE = 4\nCUFFT_INTERNAL_ERROR = 5\nCUFFT_EXEC_FAILED = 6\nCUFFT_SETUP_FAILED = 7\nCUFFT_INVALID_SIZE = 8\nCUFFT_INCOMPLETE_PARAMETER_LIST = 10\n HIPFFT_INVALID_DEVICE = 11\nCUFFT_PARSE_ERROR = 12\nCUFFT_NO_WORKSPACE = 13\n",res);	
	exit(-1);
	}
	for(i=0;i<ntrace;i++)
	{

		copy_complex_matrix_shift<<<bk,th>>>(dev_s_mat,dev_dat+i*new_npts,new_npts);
		//hipDeviceSynchronize();
	//w*s
		complex_mul_float_mat<<<blocks,threads>>>(dev_s_mat,dev_w_mat,new_npts);	
		//hipDeviceSynchronize();
		//ifft
		for(j=0;j<nloop;j++)
		{
                	hipfftExecC2C(plan2,dev_s_mat+j*shift,dev_s_mat+j*shift,HIPFFT_BACKWARD);
			//hipDeviceSynchronize();
		}
		//caution:
		//s[freq][time], so ft_mat[freq][time] is ok
		//new kernel function
		//
		update_smat<<<blocks,threads>>>(dev_s_mat,dev_ft_mat,new_npts);
		//hipDeviceSynchronize();

		//new kernel function
		if(i!=0)
		{
			sum_smat<<<blocks,threads>>>(dev_sum_s,dev_s_mat,new_npts);
		//hipDeviceSynchronize();
		}
		else
		{
			copy_complex_matrix<<<blocks,threads>>>(dev_sum_s,dev_s_mat,new_npts);
		//hipDeviceSynchronize();
		}

	}
	//final weight
	//new kernel function
	pws_wght_mat<<<blocks,threads>>>(dev_sum_s,new_npts,ntrace,pwr);
		//hipDeviceSynchronize();
	t5=clock();
	//linear stacking stran
	res=hipfftPlan1d(&plan0,new_npts,HIPFFT_C2C,1);
	if(res!=0)
	{
	fprintf(stderr,"fail to create  plan0, error # %d\nCUFFT_INVALID_PLAN = 1\nCUFFT_ALLOC_FAILED = 2\nCUFFT_INVALID_TYPE = 3\nCUFFT_INVALID_VALUE = 4\nCUFFT_INTERNAL_ERROR = 5\nCUFFT_EXEC_FAILED = 6\nCUFFT_SETUP_FAILED = 7\nCUFFT_INVALID_SIZE = 8\nCUFFT_INCOMPLETE_PARAMETER_LIST = 10\n HIPFFT_INVALID_DEVICE = 11\nCUFFT_PARSE_ERROR = 12\nCUFFT_NO_WORKSPACE = 13\n",res);	
	exit(-1);
	}
//new one
		hipMemcpy(dev_stack,host_stack,sizeof(hipfftComplex)*new_npts,hipMemcpyHostToDevice);	
	hipfftExecC2C(plan0,dev_stack,dev_stack,HIPFFT_FORWARD);
	hipMemcpy(host_stack,dev_stack,sizeof(hipfftComplex)*new_npts,hipMemcpyDeviceToHost);	
	//stran part
	copy_complex_matrix_shift<<<bk,th>>>(dev_s_mat,dev_stack,new_npts);
	//hipDeviceSynchronize();

	//do W*S
	complex_mul_float_mat<<<blocks,threads>>>(dev_s_mat,dev_w_mat,new_npts);
	//hipDeviceSynchronize();
	//ifft
	for(j=0;j<nloop;j++)
	{
                hipfftExecC2C(plan2,dev_s_mat+j*shift,dev_s_mat+j*shift,HIPFFT_BACKWARD);
		//hipDeviceSynchronize();
	}
	hipMemcpy(s_mat,dev_s_mat,sizeof(hipfftComplex)*new_npts*new_npts,hipMemcpyDeviceToHost);
	//end output s stran fo stack
	t6=clock();
	//now we have stran of stacking trace in s_mat
	//hipfftDestroy(plan2);

	//stran_inv
	//new kernel function
	mul_wght_mat<<<blocks,threads>>>(dev_s_mat,dev_sum_s,new_npts);
		//hipDeviceSynchronize();
	//very slow!!!
	//	sum_along_col<<<64,new_npts/64>>>(dev_stack,dev_s_mat,new_npts);
	
	hipMemcpy(s_mat,dev_s_mat,sizeof(hipfftComplex)*new_npts*new_npts,hipMemcpyDeviceToHost);
	//integrate over time axis to get sp
	for(i=0;i<new_npts;i++)//freq
	{
		host_stack[i].x=0.0;
		host_stack[i].y=0.0;
		for(j=0;j<new_npts;j++)//time
		{
		host_stack[i].x=host_stack[i].x+s_mat[i*new_npts+j].x;
		host_stack[i].y=host_stack[i].y+s_mat[i*new_npts+j].y;
		}
		if(isnan(hipCabsf(host_stack[i]))){host_stack[i].x=0;host_stack[i].y=0;}
	}
	host_stack[0].x=0.0;
	host_stack[0].y=0.0;

	//ifft
	//new one
	hipMemcpy(dev_stack,host_stack,sizeof(hipfftComplex)*new_npts,hipMemcpyHostToDevice);
	hipfftExecC2C(plan0,dev_stack,dev_stack,HIPFFT_BACKWARD);
	//hipDeviceSynchronize();
//new one
	hipMemcpy(host_stack,dev_stack,sizeof(hipfftComplex)*new_npts,hipMemcpyDeviceToHost);
	t7=clock();
	//output time costs

	time_cost=((double)(t2-t1))/CLOCKS_PER_SEC;
	printf("readding files takes %f s\n",time_cost);
	time_cost=((double)(t4-t3))/CLOCKS_PER_SEC;
	printf("FFT of all files takes %f s\n",time_cost);
	time_cost=((double)(t5-t4))/CLOCKS_PER_SEC;
	printf("Stran of all files takes %f s\n",time_cost);
	time_cost=((double)(t6-t5))/CLOCKS_PER_SEC;
	printf("Stran of single file takes ~ %f s\n",time_cost);
	time_cost=((double)(t7-t1))/CLOCKS_PER_SEC;
	printf("In total %f s\n",time_cost);
	//free plans
	hipfftDestroy(plan0);
	hipfftDestroy(plan1);
	hipfftDestroy(plan2);
	//free other cuda 
	hipFree(dev_stack);
	hipFree(dev_dat);
	hipFree(dev_s_mat);
	hipFree(dev_w_mat);
	hipFree(dev_ft_mat);
	hipFree(dev_tmp);
	hipFree(dev_sum_s);
	hipHostFree(host_dat);
	//hipHostFree(host_stack);
	//only real part is useful
	SACHEAD hd = sachdr(dt,new_npts,b);
	float *tmp=(float*)malloc(sizeof(float)*new_npts);
	for(i=0;i<new_npts;i++)
	{
		tmp[i]=host_stack[i].x/new_npts;
		if(isnan(tmp[i])) tmp[i]=0.0;
	}
	write_sac(av[2],hd,tmp);
	hipHostFree(host_stack);
}
void mul_wght(hipfftComplex *a,hipfftComplex *b,int n)
{
	int i;
	float tmpx,tmpy;
	for(i=0;i<n;i++)
	{
		tmpx=a[i].x*b[i].x;
		tmpy=a[i].y*b[i].x;
		a[i].x=tmpx;
		a[i].y=tmpy;
	}
}
void pws_wght(hipfftComplex *s,int n,int ntr,int pwr)
{
	int i;
	float abs;
	for(i=0;i<n;i++)
	{
		abs=sqrt(s[i].x*s[i].x + s[i].y*s[i].y);
		s[i].x = abs*abs;
		}	
}
void sum_s_mat(hipfftComplex *sum,hipfftComplex *s,int n)
{
	int i;
	for(i=0;i<n;i++)
	{
		sum[i].x=(sum[i].x + s[i].x);	
		sum[i].y=(sum[i].y + s[i].y);	
	}
}
void update_s_mat(hipfftComplex *s,hipfftComplex *ft,int n)
{
//  s/abs(s) * exp(2*pi*i*fvec*tvec)
	int i;
	float abs;
	float tmpx,tmpy;
	for(i=0;i<n;i++)
	{
		abs=sqrt(s[i].x*s[i].x + s[i].y*s[i].y);
		if(isnan(abs)) abs=1;
		tmpx=(s[i].x * ft[i].x - s[i].y*ft[i].y)/abs;	
		tmpy=(s[i].y * ft[i].x + s[i].x*ft[i].y)/abs;	
		s[i].x=tmpx;
		s[i].y=tmpy;
	}
}

//return 2^n >= a
int nextpow2(int a)
{
        int x;
	x=1;
        while(x<a)
        {
                x=x*2;		
	}
        return(x);
}
void gaussian_matrix(float *w,int k,float *fvec,int n)
{
        int i,j;
	float a,freq;
        float pi=3.14159;
	for(i=0;i<n;i++)
	{
	freq=fvec[i];
	for(j=0;j<n;j++)
	{
		a=fvec[j];
		w[i*n+j]=exp(-2*pi*pi*a*a/((k/2)*freq*freq));
	}
	}

}
void complex_mul_float(hipfftComplex *a,float *b,int n)
{
        int i;
	for(i=0;i<n;i++)
	{
		a[i].x=a[i].x*b[i];
		a[i].y=a[i].y*b[i];
	}
}

void read_trace_list(char *list,float **dat,int *n,int *npts,float *dt,float *b)
{
	FILE *fp;
	SACHEAD hd0,hd1;
	
	int i;
	char line[200];
	char name[100];
	fp=fopen(list,"r");
	fgets(line,99,fp);
	sscanf(line,"%s",name);
	dat[0]=read_sac(name,&hd0);

	(*b) = hd0.b;
	(*dt) = hd0.delta;
	(*npts) = hd0.npts;
	i = 1;

	while(fgets(line,99,fp) != NULL)
	{
		sscanf(line,"%s",name);
		dat[i]=read_sac(name,&hd1);
		i=i+1;
		if(hd1.delta != hd0.delta || hd1.npts != hd0.npts)
		{
			fprintf(stderr,"Different dt or npts %s\n",name);
			i=i-1; 
		}
//		fprintf(stderr,"read %s\n",name);
	}
	(*n)=i;
}
